#include "hip/hip_runtime.h"
#include "array_indexer.cuh"

__global__ void array1d(float  *inputArray, size_t sizeofArray)
{
    threadIdx.x;
}

void array1d_wrapper(float  *inputArray, size_t sizeofArray)
{
	int blockSize = 256;
	int numBlocks = (N + blockSize - 1) / blockSize;
    array1d<<<numBlocks, blockSize>>> (inputArray,sizeofArray);
}