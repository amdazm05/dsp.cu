#include "hip/hip_runtime.h"
#include "array_indexer.cuh"

inline void _array1d(float  *inputArray, size_t sizeofArray,dim3  _threads_block,dim3 _blocks_grid)
{
    array1d<<<_blocks_grid,_threads_block>>>(inputArray,sizeofArray);
}

__global__ void array1d(float  *inputArray, size_t sizeofArray)
{
    int index = threadIdx.x; 
    printf("Data Indexed here : x%d y%d , array [%f]\n",threadIdx.x,threadIdx.y,inputArray[index]);
}
