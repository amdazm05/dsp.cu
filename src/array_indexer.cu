
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#include <iostream>

__global__ void array1d(float  *inputArray, size_t sizeofArray)
{
    int index = threadIdx.x; 
    printf("Data Indexed here : x%d  , array [%f]\n",threadIdx.x,inputArray[index]);
}

__global__ void array2d(float  *inputArray, size_t sizeofArray)
{
    int index = threadIdx.x +threadIdx.x*threadIdx.y; 
    printf("Data Indexed here : x%d y%d , array [%f]\n",threadIdx.x,threadIdx.y,inputArray[index]);
}

__global__ void array3d(float  *inputArray, size_t sizeofArray)
{
    int index = threadIdx.x +threadIdx.x*threadIdx.y + blockDim.x*blockDim.y*threadIdx.z; 
    printf("Data Indexed here : x%d y%d z%d , array [%f]\n",threadIdx.x,threadIdx.y,threadIdx.z,inputArray[index]);
}

inline void _array1d(float  *inputArray, size_t sizeofArray,dim3  _threads_block,dim3 _blocks_grid)
{
    array1d<<<_blocks_grid,_threads_block>>>(inputArray,sizeofArray);
}
inline void  _array2d(float  *inputArray, size_t sizeofArray,dim3  _threads_block,dim3  _blocks_grid)
{
    array2d<<<_blocks_grid,_threads_block>>>(inputArray,sizeofArray);
}